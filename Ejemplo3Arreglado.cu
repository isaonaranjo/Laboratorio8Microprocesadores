// Mar�a Isabel Ortiz Naranjo
#include <stdio.h>  // le agregu� el #
#include <stdlib.h> 
#include <hip/hip_runtime.h>

#define N 16 

__global__ void kernel( int *a, int *b, int *c ) // Agregu� *b
{
	int myID = threadIdx.x + blockDim.x * blockIdx.x;
	// Solo trabajan N hilos
	if (myID < N)
	{
		c[myID] = a[myID] + b[myID];
	}
}
__global__ void kernel2( int *a, int *b, int *c )
{
	// Originalmente no funcionaba, ya que faltaba el Id del bloque a utilizar
	int myID = threadIdx.x + blockDim.x* blockIdx.x;

	// Solo trabajan N hilos
	if (myID < N)
	{
		c[myID] = a[myID] * b[myID];
	}
}
int main(int argc, char** argv)
{
	hipStream_t stream1, stream2;
	
	int *a1, *b1, *c1; 									// stream 1 mem ptrs
	int *a2, *b2, *c2; 									// stream 2 mem ptrs
	int *dev_a1, *dev_b1, *dev_c1; 						// stream 1 mem ptrs
	int *dev_a2, *dev_b2, *dev_c2; 						// stream 2 mem ptrs
	
	//stream 1
	hipMalloc( (void**)&dev_a1, N * sizeof(int) );
	hipMalloc( (void**)&dev_b1, N * sizeof(int) );
	hipMalloc( (void**)&dev_c1, N * sizeof(int) );

	hipHostAlloc( (void**)&a1, N * sizeof(int), hipHostMallocDefault);
	hipHostAlloc( (void**)&b1, N * sizeof(int), hipHostMallocDefault);
	hipHostAlloc( (void**)&c1, N * sizeof(int), hipHostMallocDefault);
	
	//stream 2
	hipMalloc( (void**)&dev_a2, N * sizeof(int) );
	hipMalloc( (void**)&dev_b2, N * sizeof(int) );
	hipMalloc( (void**)&dev_c2, N * sizeof(int) );

	hipHostAlloc( (void**)&a2, N * sizeof(int), hipHostMallocDefault);
	hipHostAlloc( (void**)&b2, N * sizeof(int), hipHostMallocDefault);
	hipHostAlloc( (void**)&c2, N * sizeof(int), hipHostMallocDefault);
	
	for (int i =0; i<N; i++){
		a1[i]= i;
		b1[i]= a1[i] + i;

		a2[i]= i;
		b2[i]= a1[i] * i;

	}

	for(int i=0;i < N;i+= N*2) { // loop over data in chunks
	// interweave stream 1 and steam 2
		
		hipMemcpyAsync(dev_a1,a1,N*sizeof(int),hipMemcpyHostToDevice,stream1); // Faltaba los Async en la memoria cuda
		hipMemcpyAsync(dev_a2,a2,N*sizeof(int),hipMemcpyHostToDevice,stream2); // Faltaba los Async en la memoria cuda
		hipMemcpyAsync(dev_b1,b1,N*sizeof(int),hipMemcpyHostToDevice,stream1); // Faltaba los Async en la memoria cuda
		hipMemcpyAsync(dev_b2,b2,N*sizeof(int),hipMemcpyHostToDevice,stream2); // Faltaba los Async en la memoria cuda
		
		kernel<<<(int)ceil(N/1024)+1,1024,0,stream1>>>(dev_a1,dev_b1,dev_c1);
		kernel2<<<(int)ceil(N/1024)+1,1024,0,stream2>>>(dev_a2,dev_b2,dev_c2);
		
		hipMemcpyAsync(c1,dev_c1,N*sizeof(int),hipMemcpyDeviceToHost,stream1);
		hipMemcpyAsync(c2,dev_c2,N*sizeof(int),hipMemcpyDeviceToHost,stream2);
	}

	hipStreamSynchronize(stream1); // Agregue Synchronize 
	hipStreamSynchronize(stream2); 
	
	printf("Stream 1 \n");
	printf("a1 \n");
	for (int i =0; i<N; i++){
		printf("%d \n",a1[i]);
	}
	printf("b1 \n");
	for (int i =0; i<N; i++){
		printf("%d \n",b1[i]);
	}
	printf("c1 \n");
	for (int i =0; i<N; i++){
		printf("%d \n",c1[i]);
	}
	printf("Stream 2 \n");
	printf("a2 \n");
	for (int i =0; i<N; i++){
		printf("%d \n",a2[i]);
	}
	printf("b2 \n");
	for (int i =0; i<N; i++){
		printf("%d \n",b2[i]);
	}
	printf("c2 \n");
	for (int i =0; i<N; i++){
		printf("%d \n",c2[i]);
	}
	hipStreamDestroy(stream1); // Agregue un Destroy
	hipStreamDestroy(stream2);

	return 0;
	
}